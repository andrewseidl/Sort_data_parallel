#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include <iostream>
#include <vector>
#include <ctime>

#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "utils.h"


int main() {
    std::vector<double> V;
    thrust::device_vector<double> d_V;
    
    //use the system time to create a random seed
    unsigned int seed = (unsigned int) time(NULL);
    
    size_t step = 10;
    size_t mem = 10000000;

    for(size_t i = 16; i <= mem; i = 2 * step, step *= 1.1) {
	    hipEvent_t start, stop, startcopy, stopcopy;
	    hipEventCreate(&startcopy);
	    hipEventCreate(&start);
	    hipEventCreate(&stop);
	    hipEventCreate(&stopcopy);

		//Fill V with random numbers in the range [0,1]:
        V.resize(i);
        rnd_fill(V, 0.0, 1.0, seed);
		hipEventRecord(startcopy,0);
        d_V = V;

	    //Start recording
	    hipEventRecord(start,0);
        
        thrust::stable_sort(d_V.begin(), d_V.end());
        
	    //Stop recording
	    hipEventRecord(stop,0);

		//Copy data back to CPU
		thrust::copy(d_V.begin(), d_V.end(), V.begin());
		hipEventRecord(stopcopy,0);

	    hipEventSynchronize(stopcopy);
	    float inclusiveTime, exclusiveTime;
	    hipEventElapsedTime(&exclusiveTime, start, stop);
	    hipEventElapsedTime(&inclusiveTime, startcopy, stopcopy);

	    hipEventDestroy(startcopy);
	    hipEventDestroy(start);
	    hipEventDestroy(stop);
	    hipEventDestroy(stopcopy);

	    std::cout << i << "\t" << exclusiveTime << "\t" << inclusiveTime<< std::endl;
    }
    
    return 0;
}

